#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <random>

// You may increase this value to test larger matrices
// But it will be slow on CPU
constexpr int MAXN = 1 << 28;

void vectorAddCPU(float *a, float *b, float *c, const int N) {
    for (int i = 0; i < N; ++i) {
        c[i] = a[i] + b[i];
    }
}

void initialize(float *a, float *b, const int N) {
    auto gen = std::mt19937(2024);
    auto dis = std::uniform_real_distribution<float>(-1.0, 1.0);
    for (int i = 0; i < N; ++i) {
        a[i] = dis(gen);
    }
    for (int i = 0; i < N; ++i) {
        b[i] = dis(gen);
    }
}

bool compare(float *a, float *b, const int N) {
    for (int i = 0; i < N; ++i) {
        if (std::abs(a[i] - b[i]) > 1e-3) {
            printf("Mismatch at index %d: %f vs %f\n", i, a[i], b[i]);
            return false;
        }
    }
    printf("Results match\n");
    return true;
}

__global__ void vectorAddGPU(float *a, float *b, float *c, const int N) {
    // Implement your vector add kernel here
    index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    float *a, *b, *c;
    a = new float[MAXN];
    b = new float[MAXN];
    c = new float[MAXN];
    initialize(a, b, MAXN);

    // CPU computation
    auto start = std::chrono::high_resolution_clock::now();
    vectorAddCPU(a, b, c, MAXN);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    printf("CPU time: %.3fs\n", elapsed.count());

    // ************** START GPU MEMORY ALLOCATION **************
    // Implement your code here
    // MAXN --size of the vector
    // allocate memory for GPU
    float *a_gpu, *b_gpu, *c_gpu;
    hipMalloc(a_gpu, sizeof(float) * MAXN);
    hipMalloc(b_qpu, sizeof(float) * MAXN);
    hipMalloc(c_gpu, sizeof(float) * MAXN);

    // copy
    hipMemcpy(a_gpu, a, MAX * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, MAX * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_gpu, c, MAX * sizeof(float), hipMemcpyHostToDevice);

    // define size of block and grid
    int threadSize = 256;
    int blockSize = (MAXN + threadSize - 1) / threadSize;
    // ************** START GPU COMPUTATION **************
    start = std::chrono::high_resolution_clock::now();
    // Implement your code here
    vectorAddGPU<<<blockSize, threadSize>>>(a_gpu, b_gpu, c_gpu, MAXN);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    float *result = new float[MAXN];
    // Copy the result from GPU to CPU
    hipMemcpy(result, c_gpu, MAXN * sizeof(float), hipMemcpyDeviceToHost);
    if (compare(c, result, MAXN)) {
        std::chrono::duration<double> new_elapsed = end - start;
        printf("GPU time: %.3fs\n", new_elapsed.count());
        printf("Speedup: %.2fx\n", elapsed.count() / new_elapsed.count());
    }
}