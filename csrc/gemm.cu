#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <random>

// You may increase this value to test larger matrices
// But it will be slow on CPU
constexpr int MAXN = 4096;

/**
 * @brief A naive implementation of matrix multiplication on CPU.
 * Perform C = A * B, where A is M x K, B is K x N, and C is M x N.
 */
void naiveSgemm(float *a, float *b, float *c, const int M, const int N,
                const int K) {
  for (int m = 0; m < M; ++m) {
    for (int n = 0; n < N; ++n) {
      float sum = 0.0;
      for (int k = 0; k < K; ++k) {
        sum += a[m * K + k] * b[k * N + n];
      }
      c[m * N + n] = sum;
    }
  }
}

/**
 * @brief A naive implementation of matrix multiplication on GPU.
 * Perform C = A * B, where A is M x K, B is K x N, and C is M x N.
 */
__global__ void mySgemm2D(float *a, float *b, float *c, const int M,
                             const int N, const int K) {
  int m = blockIdx.x * blockDim.x + threadIdx.x; // Row index
  int n = blockIdx.y * blockDim.y + threadIdx.y; // Column index
  __shared__ float shareA[16][16];
  __shared__ float shareB[16][16];

  float Val = 0.0;

  for (int i = 0; i < (K + 16 - 1) / 16; i++) {
      sharedA[threadIdx.y][threadIdx.x] = a[n * K + i * 16 + threadIdx.x];
      sharedB[threadIdx.y][threadIdx.x] = b[m + (threadIdx.y + i * 16) * K];
      __syncthreads();
      for (int k = 0; k < 16; k++) {
          Val += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
      }
      __syncthreads();
  }
  c[n * K + m] = Val;
}

/**
 * @brief Launch naiveSgemm2D kernel.
 */
void launchSgemm2D(float *a, float *b, float *c, const int M, const int N,
                   const int K) {
  dim3 block(16, 16); // 256 threads per block (16 * 16 = 256)
  dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);
  mySgemm2D<<<grid, block>>>(a, b, c, M, N, K);
}

void initialize(float *a, float *b, float *c, const int M, const int N,
                const int K) {
  auto gen = std::mt19937(2024);
  auto dis = std::uniform_real_distribution<float>(-1.0, 1.0);
  for (int i = 0; i < M * K; ++i) {
    a[i] = dis(gen);
  }
  for (int i = 0; i < K * N; ++i) {
    b[i] = dis(gen);
  }
  for (int i = 0; i < M * N; ++i) {
    c[i] = 0.0;
  }
}

/** 
 * @brief Launch sgemm using cuBLAS
 */
void launchCublasSgemm(float *a, float *b, float *c, const int M, const int N,
                       const int K) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0;
  float beta = 0.0;
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b, N, a, K,
              &beta, c, N);
}


int main() {
  float *a, *b, *c;
  a = new float[MAXN * MAXN];
  b = new float[MAXN * MAXN];
  c = new float[MAXN * MAXN];
  initialize(a, b, c, MAXN, MAXN, MAXN);

  // ********** CPU **********
  auto start = std::chrono::high_resolution_clock::now();
  naiveSgemm(a, b, c, MAXN, MAXN, MAXN);
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  printf("CPU time: %.3fs\n", elapsed.count());

  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, MAXN * MAXN * sizeof(float));
  hipMalloc(&d_b, MAXN * MAXN * sizeof(float));
  hipMalloc(&d_c, MAXN * MAXN * sizeof(float));
  hipMemcpy(d_a, a, MAXN * MAXN * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, MAXN * MAXN * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, MAXN * MAXN * sizeof(float), hipMemcpyHostToDevice);

  // ********** GPU **********
  start = std::chrono::high_resolution_clock::now();
  launchSgemm2D(d_a, d_b, d_c, MAXN, MAXN, MAXN);
  hipDeviceSynchronize();
  end = std::chrono::high_resolution_clock::now();
  elapsed = end - start;
  printf("GPU time: %.3fs\n", elapsed.count());

  // ********** cuBLAS **********
  start = std::chrono::high_resolution_clock::now();
  launchCublasSgemm(d_a, d_b, d_c, MAXN, MAXN, MAXN);
  hipDeviceSynchronize();
  end = std::chrono::high_resolution_clock::now();
  elapsed = end - start;
  printf("cuBLAS time: %.3fs\n", elapsed.count());
}
